/*
This is the central piece of code. This file implements a class
(interface in manager.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include <chrono>
#include <thread>
using namespace std;

GPUTransformer::GPUTransformer (float* point_host_, int size_, int* x_, int* y_, int* height_, int max_length_, int max_height_, int num_x_, int num_y_, int num_height_, int enough_large_) {
  point_host = point_host_;
  h_max_length = max_length_;
  h_max_height_ = max_height_;
  h_num_height = num_height_;
  enough_large = enough_large_;
  
  // grid size in x and y dir
  h_num_x = num_x_;
  h_num_y = num_y_;

  size = size_* 3 * sizeof(float);
  d_size = size_;

  hipMalloc((void**) &point_device, size);
  hipMalloc((void**) &x, d_size * sizeof(int));
  hipMalloc((void**) &y, d_size * sizeof(int));
  hipMalloc((void**) &height, d_size * sizeof(int));
  
  hipMemcpy(point_device, point_host, size, hipMemcpyHostToDevice);
  hipMemcpy(y, y_, d_size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(height, height_, d_size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(x, x_, d_size * sizeof(int), hipMemcpyHostToDevice);
}


void GPUTransformer::transform() {
  dim3 blockSize(256);
  dim3 gridSize((d_size + blockSize.x - 1) / blockSize.x);
  point2gridmap<<<gridSize, blockSize>>>(point_device, x, y, height, d_size, h_max_length, h_max_height_, h_num_x, h_num_y, h_num_height);
  hipDeviceSynchronize();
}


void GPUTransformer::retreive(float* point_transformed) {
  int x_h[d_size] = {0};
  int y_h[d_size] = {0};
  int height_h[d_size] = {0};

  hipMemcpy(x_h, x, d_size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(y_h, y, d_size * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(height_h, height, d_size * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < d_size; i++)
  {
    point_transformed[3*(y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x) + 0] = point_host[i];
    point_transformed[3*(y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x) + 1] = point_host[i + d_size];
    // point_transformed[3*(y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x) + 2] = point_host[i + 2 * d_size];
    point_transformed[3*(y_h[i] + x_h[i] * h_num_y + height_h[i] * h_num_y * h_num_x) + 2] = 1;
  }

  hipFree(point_device);
  hipFree(height);
  hipFree(y);
  hipFree(x);
}


GPUTransformer::~GPUTransformer() {
  hipFree(point_device);
  hipFree(height);
  hipFree(y);
  hipFree(x);
}
